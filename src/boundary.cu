#include "hip/hip_runtime.h"
#include <sph_boundary_particles/boundary.cuh>

extern "C"
{
	unsigned int iDivUp(unsigned int a, unsigned int b)
	{
		return (a % b != 0) ? (a / b + 1) : (a / b);
	}

	void computeGridSize(unsigned int n, unsigned int blockSize, unsigned int &numBlocks, unsigned int &numThreads)
	{
		numThreads = min(blockSize, n);
		numBlocks = iDivUp(n, numThreads);
	}

	void updateVbi(float* boundary_pos, float* vbi, float ir, unsigned int num_boundaries)
	{
		float* d_boundary_pos, * d_vbi;

		hipMalloc((void**)&d_boundary_pos, num_boundaries*4*sizeof(float));
		hipMalloc((void**)&d_vbi, num_boundaries*sizeof(float));

		//hipMemcpy
		hipMemcpy(d_vbi, vbi, num_boundaries*sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(boundary_pos, d_boundary_pos, num_boundaries*sizeof(float)*4, hipMemcpyHostToDevice);

		//kernel call
		
		unsigned int numThreads, numBlocks;
		computeGridSize(num_boundaries, 256, numBlocks, numThreads);

		computeVbi<<<numBlocks, numThreads>>>((float4*)boundary_pos, d_vbi, ir,num_boundaries);

		//transfer back to host mem
		hipMemcpy(vbi, d_vbi, num_boundaries*sizeof(float), hipMemcpyDeviceToHost);

		//hipFree
		hipFree(d_boundary_pos);
		hipFree(d_vbi);
	}
}
