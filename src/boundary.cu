#include "hip/hip_runtime.h"
#include "boundary.cuh"

#include "boundary_kernel.cuh"

extern "C"
{
	unsigned int iDivU(unsigned int a, unsigned int b)
	{
		return (a % b != 0) ? (a / b + 1) : (a / b);
	}

	void computeGridSiz(unsigned int n, unsigned int blockSize, unsigned int &numBlocks, unsigned int &numThreads)
	{
		numThreads = min(blockSize, n);
		numBlocks = iDivU(n, numThreads);
	}

	void updateVbi(float* boundary_pos, float* vbi, float ir, unsigned int num_boundaries)
	{
		float* d_boundary_pos;
		float* d_vbi;

		hipMalloc((void**)&d_boundary_pos, num_boundaries*4*sizeof(float));
		hipMalloc((void**)&d_vbi, num_boundaries*sizeof(float));

		printf("boundary_pos = %8f %8f %8f\n", boundary_pos[0], boundary_pos[1], boundary_pos[2]);

		//hipMemcpy
		hipMemcpy(d_vbi, vbi, num_boundaries*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(d_boundary_pos, boundary_pos, num_boundaries*sizeof(float)*4, hipMemcpyHostToDevice);

		/*//kernel call*/
		
		unsigned int numThreads, numBlocks;
		computeGridSiz(num_boundaries, 256, numBlocks, numThreads);

		computeVbi<<<numBlocks, numThreads>>>((float4*)boundary_pos, d_vbi, ir,num_boundaries);

		/*//transfer back to host mem*/
		hipMemcpy(vbi, d_vbi, num_boundaries*sizeof(float), hipMemcpyDeviceToHost);

		/*//hipFree*/
		hipFree(d_boundary_pos);
		hipFree(d_vbi);
	}
}
